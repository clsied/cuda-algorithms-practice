#include <iostream>
#include <hip/hip_runtime.h>
using namespace std;

__global__
void matrix_conv_naive_kernel(const float* input, const float* kernel, float* output, int H, int W, int KH, int KW){
    
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    int padh = KH / 2;
    int padw = KW / 2;

    float sum = 0.0f ; int ii, jj;

    if (col < W && row < H) {
        for(int ki = 0; ki < KH; ++ki){
            for(int kj = 0; kj < KW; ++kj){
                ii = row + (ki - padh);
                jj = col + (kj - padw);

                if(ii >= 0 && ii < H && jj >= 0 && jj < W){
                    sum += input[ii * W + jj] * kernel[ki * KH + kj];
                }
            }
        }
        output[row * W + col] = sum;
    }

}


void matrix_conv_naive_gpu(const float* h_input, const float* h_kernel, float* h_output, int H, int W, int KH, int KW){
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    float *d_input, *d_kernel, *d_output;
    
    hipMalloc(&d_input, sizeof(float) * H * W);
    hipMalloc(&d_kernel, sizeof(float) * KH * KW);
    hipMalloc(&d_output, sizeof(float) * H * W);

    hipMemcpy(d_input, h_input, sizeof(float) * H * W, hipMemcpyHostToDevice);
    hipMemcpy(d_kernel, h_kernel, sizeof(float) * KH * KW, hipMemcpyHostToDevice);

    dim3 blockDim(16, 16);
    dim3 gridDim((W + 15) / 16, (H + 15) / 16);

    hipEventRecord(start);
    matrix_conv_naive_kernel<<<gridDim, blockDim>>>(d_input, d_kernel, d_output, H, W, KH, KW);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    cout << "GPU (naive) time: " << ms << " ms\n";

    hipMemcpy(h_output, d_output, sizeof(float) * H * W, hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_kernel);


}
