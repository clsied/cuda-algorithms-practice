#include <iostream>
#include <hip/hip_runtime.h>

__global__
void vector_add_kernel(const float* A, const float* B, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N)
        C[idx] = A[idx] + B[idx];
}

void vector_add_gpu(const float* h_A, const float* h_B, float* h_C, int N){
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    
    float *d_A, *d_B, *d_C; // pointer
    
    hipMalloc(&d_A, sizeof(float) * N); // allocate memory on GPU, input agruments is address of pointer
    hipMalloc(&d_B, sizeof(float) * N);
    hipMalloc(&d_C, sizeof(float) * N);
    
    hipMemcpy(d_A, h_A, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float) * N, hipMemcpyHostToDevice);
    
    const int blockSize = 256;
    
    // compute the number of grid requires,
    // addition of blockSize - 1 is equivalent to ceil
    const int gridSize = (N + blockSize - 1) / blockSize; 
    
    
    // only include the time of the kernel execution
    hipEventRecord(start);
    vector_add_kernel<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    
    
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    std::cout << "GPU time: " << ms << " ms\n";

    hipMemcpy(h_C, d_C, N * sizeof(float), hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}