#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "dense_to_sparse.h"
#define blockSize 256

using namespace std;
__global__ 
void spmv_ell_kernel(const int* col_indices, const float* vals, int rows, int max_non_zero, const float* B, float* C){

    int row = blockIdx.x * blockDim.x + threadIdx.x;
    if (row < rows){
        float sum = 0.0f;
        for (int i = 0; i < max_non_zero; i++) {
            int col = col_indices[row * max_non_zero + i];
            float val = vals[row * max_non_zero + i];
            
            if (col != -1) {
                sum += val * B[col];
            }
        }
        C[row] = sum;
    }

}
void spmv_ell_gpu(const float* h_A, const float* h_B, float* h_C, int M, int K){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    ELL h_ell = dense_to_ell(h_A, M, K);
    int *d_col_indices;
    float *d_vals, *d_B, *d_C;

    const int max_non_zero = h_ell.max_non_zero;
    const int rows = h_ell.rows;

    hipMalloc(&d_col_indices, sizeof(int) * (rows * max_non_zero));
    hipMalloc(&d_vals, sizeof(float) * (rows * max_non_zero));
    hipMalloc(&d_B, sizeof(float) * K);
    hipMalloc(&d_C, sizeof(float) * M);

    hipMemcpy(d_col_indices, h_ell.col_indices.data(), sizeof(int) *  (rows * max_non_zero), hipMemcpyHostToDevice);
    hipMemcpy(d_vals, h_ell.vals.data(), sizeof(float) *  (rows * max_non_zero), hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float) * K, hipMemcpyHostToDevice);
    // initialize to 0
    hipMemset(d_C, 0, sizeof(float) * M);
    
    hipEventRecord(start);
    spmv_ell_kernel<<< (rows + blockSize - 1) / blockSize, blockSize>>>(d_col_indices, d_vals, rows, max_non_zero, d_B, d_C);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    cout << "GPU (ELL) time: " << ms << " ms\n";

    hipMemcpy(h_C, d_C, sizeof(float) * M, hipMemcpyDeviceToHost);

    hipFree(d_col_indices);
    hipFree(d_vals);
    hipFree(d_B);
    hipFree(d_C);

}