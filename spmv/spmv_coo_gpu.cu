#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "dense_to_sparse.h"
#define blockSize 256

using namespace std;
__global__ 
void spmv_coo_kernel(const int* row, const int* col, const float* val, int non_zeros, const float* B, float* C){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < non_zeros) {
        // avoid bank conflicts
        atomicAdd(&C[row[idx]], val[idx] * B[col[idx]]);
    }
}
void spmv_coo_gpu(const float* h_A, const float* h_B, float* h_C, int M, int K){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    COO h_coo = dense_to_coo(h_A, M, K);
    int *d_row, *d_col;
    float *d_val, *d_B, *d_C;

    const int non_zeros = h_coo.val.size();

    hipMalloc(&d_row, sizeof(int) * non_zeros);
    hipMalloc(&d_col, sizeof(int) * non_zeros);
    hipMalloc(&d_val, sizeof(float) * non_zeros);
    hipMalloc(&d_B, sizeof(float) * K);
    hipMalloc(&d_C, sizeof(float) * M);

    hipMemcpy(d_row, h_coo.row.data(), sizeof(int) * non_zeros, hipMemcpyHostToDevice);
    hipMemcpy(d_col, h_coo.col.data(), sizeof(int) * non_zeros, hipMemcpyHostToDevice);
    hipMemcpy(d_val, h_coo.val.data(), sizeof(float) * non_zeros, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float) * M, hipMemcpyHostToDevice);
    // initialize to 0
    hipMemset(d_C, 0, sizeof(float) * K);
    
    hipEventRecord(start);
    spmv_coo_kernel<<< (non_zeros + blockSize - 1) / blockSize, blockSize>>>(d_row, d_col, d_val, non_zeros, d_B, d_C);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    cout << "GPU (COO) time: " << ms << " ms\n";

    hipMemcpy(h_C, d_C, sizeof(float) * M, hipMemcpyDeviceToHost);

    hipFree(d_row);
    hipFree(d_col);
    hipFree(d_val);
    hipFree(d_B);
    hipFree(d_C);

}