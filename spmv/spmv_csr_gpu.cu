#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include "dense_to_sparse.h"
#define blockSize 256

using namespace std;
__global__ 
void spmv_csr_kernel(const int* row_ptr, const int* col_idx, const float* val, int rows, const float* B, float* C){
    
    int row = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < rows) {
        float sum = 0.0f;
        int start = row_ptr[row];
        int end   = row_ptr[row + 1];
        for (int i = start; i < end; ++i) {
            sum += val[i] * B[col_idx[i]];
        }
        C[row] = sum;
    }
}
void spmv_csr_gpu(const float* h_A, const float* h_B, float* h_C, int M, int K){
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    CSR h_csr = dense_to_csr(h_A, M, K);
    int *d_row_ptr, *d_col_idx;
    float *d_val, *d_B, *d_C;

    const int non_zeros = h_csr.col_idx.size();
    const int rows = h_csr.row_ptr.size() - 1;

    hipMalloc(&d_row_ptr, sizeof(int) * (rows + 1));
    hipMalloc(&d_col_idx, sizeof(int) * non_zeros);
    hipMalloc(&d_val, sizeof(float) * non_zeros);
    hipMalloc(&d_B, sizeof(float) * K);
    hipMalloc(&d_C, sizeof(float) * M);


    hipMemcpy(d_row_ptr, h_csr.row_ptr.data(), sizeof(int) * (rows + 1), hipMemcpyHostToDevice);
    hipMemcpy(d_col_idx, h_csr.col_idx.data(), sizeof(int) * non_zeros, hipMemcpyHostToDevice);
    hipMemcpy(d_val, h_csr.val.data(), sizeof(float) * non_zeros, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, sizeof(float) * K, hipMemcpyHostToDevice);
    
    // initialize to 0
    hipMemset(d_C, 0, sizeof(float) * M);
    
    hipEventRecord(start);
    // each thread map to a row, to avoid the use of atomic operation
    spmv_csr_kernel<<< (rows + blockSize - 1) / blockSize, blockSize>>>(d_row_ptr, d_col_idx, d_val, rows, d_B, d_C);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    
    float ms = 0;
    hipEventElapsedTime(&ms, start, stop);
    cout << "GPU (CSR) time: " << ms << " ms\n";

    hipMemcpy(h_C, d_C, sizeof(float) * M, hipMemcpyDeviceToHost);

    hipFree(d_row_ptr);
    hipFree(d_col_idx);
    hipFree(d_val);
    hipFree(d_B);
    hipFree(d_C);

}